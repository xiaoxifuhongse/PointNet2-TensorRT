#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <stdio.h>
extern "C"
//Adds two arrays
   void runCudaPart(std::vector<inno_point>&pts);



__device__ void __update(float *__restrict__ dists, int *__restrict__ dists_i,
                         int idx1, int idx2) {
  const float v1 = dists[idx1], v2 = dists[idx2];
  const int i1 = dists_i[idx1], i2 = dists_i[idx2];
  dists[idx1] = max(v1, v2);
  dists_i[idx1] = v2 > v1 ? i2 : i1;
}

// Input dataset: (b, n, 3), tmp: (b, n)
// Ouput idxs (b, m)
template <unsigned int block_size>
__global__ void furthest_point_sampling_kernel(
    int b, int n, int m, const float *__restrict__ dataset,
    float *__restrict__ temp, int *__restrict__ idxs) {
  if (m <= 0) return;


  __shared__ float dists[block_size];
  __shared__ int dists_i[block_size];

  int batch_index = blockIdx.x;
  dataset += batch_index * n * 3;
  temp += batch_index * n;
  idxs += batch_index * m;

  int tid = threadIdx.x;
  const int stride = block_size;

  int old = 0;
  if (threadIdx.x == 0) idxs[0] = old;

  __syncthreads();
  for (int j = 1; j < m; j++) {
    int besti = 0;
    float best = -1;
    float x1 = dataset[old * 3 + 0];
    float y1 = dataset[old * 3 + 1];
    float z1 = dataset[old * 3 + 2];
    for (int k = tid; k < n; k += stride) {
      float x2, y2, z2;
      x2 = dataset[k * 3 + 0];
      y2 = dataset[k * 3 + 1];
      z2 = dataset[k * 3 + 2];
     // printf("%f,%f,%f\n",x2,y2,z2);
      float mag = (x2 * x2) + (y2 * y2) + (z2 * z2);
      if (mag <= 1e-3) continue;

      float d =
          (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);

      float d2 = min(d, temp[k]);
      temp[k] = d2;
      besti = d2 > best ? k : besti;
      best = d2 > best ? d2 : best;
    }
    dists[tid] = best;
    dists_i[tid] = besti;
    __syncthreads();

    if (block_size >= 512) {
      if (tid < 256) {
        __update(dists, dists_i, tid, tid + 256);
      }
      __syncthreads();
    }
    if (block_size >= 256) {
      if (tid < 128) {
        __update(dists, dists_i, tid, tid + 128);
      }
      __syncthreads();
    }
    if (block_size >= 128) {
      if (tid < 64) {
        __update(dists, dists_i, tid, tid + 64);
      }
      __syncthreads();
    }
    if (block_size >= 64) {
      if (tid < 32) {
        __update(dists, dists_i, tid, tid + 32);
      }
      __syncthreads();
    }
    if (block_size >= 32) {
      if (tid < 16) {
        __update(dists, dists_i, tid, tid + 16);
      }
      __syncthreads();
    }
    if (block_size >= 16) {
      if (tid < 8) {
        __update(dists, dists_i, tid, tid + 8);
      }
      __syncthreads();
    }
    if (block_size >= 8) {
      if (tid < 4) {
        __update(dists, dists_i, tid, tid + 4);
      }
      __syncthreads();
    }
    if (block_size >= 4) {
      if (tid < 2) {
        __update(dists, dists_i, tid, tid + 2);
      }
      __syncthreads();
    }
    if (block_size >= 2) {
      if (tid < 1) {
        __update(dists, dists_i, tid, tid + 1);
      }
      __syncthreads();
    }

    old = dists_i[0];
    if (tid == 0) idxs[j] = old;
  }
}

void furthest_point_sampling_kernel_wrapper(int b, int n, int m,
                                             const float *dataset, float *temp,
                                            int *idxs) {
  unsigned int n_threads = opt_n_threads(n);

 // hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (n_threads) {
    case 512:
      furthest_point_sampling_kernel<512>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 256:
      furthest_point_sampling_kernel<256>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 128:
      furthest_point_sampling_kernel<128>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 64:
      furthest_point_sampling_kernel<64>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 32:
      furthest_point_sampling_kernel<32>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 16:
      furthest_point_sampling_kernel<16>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 8:
      furthest_point_sampling_kernel<8>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 4:
      furthest_point_sampling_kernel<4>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 2:
      furthest_point_sampling_kernel<2>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    case 1:
      furthest_point_sampling_kernel<1>
          <<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
      break;
    default:
      furthest_point_sampling_kernel<512>
          <<<b, n_threads >>>(b, n, m, dataset, temp, idxs);
  }

  CUDA_CHECK_ERRORS();
}


// input: points(b, c, n) idx(b, m)
// output: out(b, c, m)
__global__ void gather_points_kernel(int b, int c, int n, int m,
                                     const float *__restrict__ points,
                                     const int *__restrict__ idx,
                                     float *__restrict__ out) {

      for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int l = blockIdx.y; l < c; l += gridDim.y) {
          for (int j = threadIdx.x; j < m; j += blockDim.x) {

            int a = idx[i * m + j];
            out[3*j+l] = points[3*a + l];
          }
        }
      }

//  for (int i = blockIdx.x; i < b; i += gridDim.x) {
//    for (int l = blockIdx.y; l < c; l += gridDim.y) {
//      for (int j = threadIdx.x; j < m; j += blockDim.x) {

//        int a = idx[i * m + j];
//        out[(i * c + l) * m + j] = points[(i * c + l) * n + a];
//      }
//    }
//  }
}

void gather_points_kernel_wrapper(int b, int c, int n, int npoints,
                                  const float *points, const int *idx,
                                  float *out) {
    printf("%d,%d,%d,%d,%d\n",b,c,n,npoints,opt_n_threads(npoints));
  gather_points_kernel<<<dim3(b, c, 1), opt_n_threads(npoints)>>>(b, c, n, npoints,points, idx, out);

  CUDA_CHECK_ERRORS();
}


void runCudaPart(std::vector<inno_point>&pts) {

    int B = 1;
    int N = pts.size();
    int M = 8000;

    float *h_points = new float[3*N];
    for(int i=0; i<N;i++)
    {
        h_points[i*3+0]=pts[i].x;
        h_points[i*3+1]=pts[i].y;
        h_points[i*3+2]=pts[i].z;

    }
    float *h_temp = new float[N];
    for(int i=0; i<N;i++)
    {
        h_temp[i]=1e10;

    }

    int * h_smplepout= new int[M];
    float * h_output= new float[3*M];

    float *d_points_sample ;
    hipMalloc((void**)&d_points_sample, 3*M*sizeof(float));
    hipMemset(d_points_sample,0,3*M*sizeof(int));

     float *d_points;
     int *d_output;
     float *d_temp;
     hipMalloc((void**)&d_output, M*sizeof(int)) ;
     hipMemset(d_output,0,M*sizeof(int));

    hipMalloc((void**)&d_points, 3*N*sizeof(float)) ;
    hipMemcpy(d_points,h_points,3*N*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc((void**)&d_temp,N*sizeof(float));
    hipMemcpy(d_temp,h_temp,N*sizeof(float),hipMemcpyHostToDevice);

   furthest_point_sampling_kernel_wrapper(B, N, M, d_points, d_temp, d_output);

   hipMemcpy(h_smplepout,d_output,M*sizeof(int),hipMemcpyDeviceToHost);
   for(int i=0; i<M; i++)
           printf("%d\n", h_smplepout[i] );

   gather_points_kernel_wrapper(B,3,N,M,d_points,d_output,d_points_sample);



   hipMemcpy(h_output,d_points_sample,3*M*sizeof(float),hipMemcpyDeviceToHost);
   for(int i=0; i<M; i++)
           printf("%f, %f, %f\n", i, h_output[3*i+0],h_output[3*i+1],h_output[3*i+2] );
   hipFree(d_points);
   hipFree(d_temp);
   hipFree(d_output);
   free(h_points);
   free(h_temp);

}



